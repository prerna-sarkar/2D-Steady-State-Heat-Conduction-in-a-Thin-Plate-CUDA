#include "hip/hip_runtime.h"
/*
CUDA program to determine the steady state heat distribution in a thin metal plate using synchronous iteration on a GPU
*/

#include <stdio.h>
#include<fstream>
#include<iomanip> //precision
#include <unistd.h> //getopt
#include <stdlib.h>  //atoi

void TempDistribution(double*, double*, int, int);

inline hipError_t HANDLE_ERROR(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}

int main(int argc, char *argv[])
{
    int numInteriorPoints=0;
    int numIterations=0;
    int opt;
    while((opt = getopt(argc,argv,"n:I:")) != -1)     // take inputs from command line arguments
    {
        switch(opt)
        {
            case 'n':
                numInteriorPoints = atoi(optarg);
                break;
            case 'I':
                numIterations = atoi(optarg);
                break;
            default:
                printf("ERROR: Usage: [-n numInteriorPoints -I numIterations"); 
                return 1; //exit program
        }
    }
    
    if(numInteriorPoints<=0 || numIterations<=0)
    {
        printf("ERROR: numInteriorPoints and numIterations cannot be <= 0\n");
        return 1;
    }

    int width = numInteriorPoints+2;
    
    int size = width * width * sizeof(double); //allocate num of bytes for the 1D array representing points on our 2D plate
    
    // declare unified memory pointers
    double* H; 
    double* G;
    
    // allocate unified memory – accessible from CPU or GPU
    hipMallocManaged(&H,size);
    hipMallocManaged(&G,size);


    // initialise temp at boundaries to values given in the problem statement
    for (int c=0; c<width; c++) //top 
    {
        if(c<round(width*0.3)||c>=round(width*0.7))
        {
            H[0 * width + c] = 20.0;
            G[0 * width + c] = 20.0;
        }
        else
        {
            H[0 * width + c] = 100.0;
            G[0 * width + c] = 100.0;
        }
    }
    
    for (int c=0; c<width; c++) //bottom
    {
        H[((width-1) * width) + c] = 20.0;
        G[((width-1) * width) + c] = 20.0;
    }
    
    for (int r=0; r< width; r++) //left
    {
        H[((r) * width) + 0] = 20.0;
        G[((r) * width) + 0] = 20.0;
    }
    
    for (int r=1; r< width; r++) //right
    {
        H[((r) * width) + (width-1)] = 20.0;
        G[((r) * width) + (width-1)] = 20.0;
    }

    for (int r = 1; r < width-1; r++)   // initialise temp at interior points to zero
    {
        for (int c = 1; c < width-1; c++)
        {
            H[r * width + c] = 0;
            G[r * width + c] = 0;
        }
    }
    
    TempDistribution(H, G, width, numIterations);

    // Free memory
    hipFree(H);
    hipFree(G);
    
    return 0;
}


__global__ void GvalueCalculate(double* H, double* G, int width)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x ;  // Calculate the column index of the G element, denote by x
    int y = blockIdx.y * blockDim.y + threadIdx.y ; // Calculate the row index of the G element, denote by y
    
    // each thread computes one element
    if(x>0 && y>0 && x<(width-1) && y<(width-1)) // check if thread lies within the plate's interior points region
    {
        int index = y*width + x; //(row number*length of row) + column number
        int left = y*width + (x-1);
        int right = y*width + (x+1);
        int up = (y-1)*width + x;
        int down = (y+1)*width + x;
        
        G[index] = 0.25*(H[up] + H[down] + H[left] + H[right]);
    }

}


void TempDistribution(double* H, double* G, int width, int numIterations)
{
    // capture start time
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));;

    // kernel invocation code
    dim3 dimBlock(32, 32); // 32*32 = 1024 threads per block
    dim3 dimGrid((width-2)/32 + 1, (width-2)/32 + 1); // blocks per grid
   
   
    for (int i=1; i<(numIterations/2)+1; i++)
    {
        GvalueCalculate << <dimGrid, dimBlock>> > (H, G, width);
        hipDeviceSynchronize();
        GvalueCalculate << <dimGrid, dimBlock>> > (G, H, width); //swapping G and H in kernel call to avoid copying within kernel
        hipDeviceSynchronize();
    }

    std::ofstream myOutFile("finalTemperatures.csv"); //output to .csv file

    for (int r = 0; r < width; r++)
    {
        for (int c = 0; c < width; c++)
        {
            myOutFile << std::setprecision(5) <<H[r * width + c];
            if (c<width-1)
                myOutFile << ",";
        }
        myOutFile << "\n";
    }

    // get stop time, and display the timing results
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float   elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Thin plate calculation took:  %3.1f milliseconds\n", elapsedTime);

    // destroy events to free memory
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
}
